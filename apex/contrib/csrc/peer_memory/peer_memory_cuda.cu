#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <list>
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime_api.h>
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if( err != hipSuccess ) {                        \
    char hostname[1024];                            \
    gethostname(hostname, 1024);                    \
    printf("%s: CUDA failure %s:%d '%s'\n",         \
         hostname,                                  \
        __FILE__,__LINE__,hipGetErrorString(err)); \
  }                                                 \
} while(0)

namespace {

constexpr int THREADS_PER_CTA = 128;

/* Basic deleter function for from_blob function.
void deleter(void* ptr)
{
    printf("deleter(ptr=%p)\n",ptr);
    hipFree(ptr);
}
*/

template<class T>
at::Tensor blob_view(T* raw_ptr, std::vector<int64_t> shape, const at::TensorOptions& options, bool channels_last)
{
    size_t size = 1;
    std::vector<int64_t> strides(shape.size());
    if (channels_last) {
        assert(shape.size() == 4);
        strides[0] = shape[1]*shape[2]*shape[3];
        strides[1] = 1;
        strides[2] = shape[1]*shape[3];
        strides[3] = shape[1];
    } else {
        int idx = strides.size();
        for (auto it = shape.rbegin();  it != shape.rend();  ++it)
        {
	    strides[--idx] = size;
	    size *= *it;
        }
    }
    size *= sizeof(T);
    // TODO: Implement dynamic reuse of pooled peer memory.
    // We provide no deleter function because all peer memory allocations are static in this implementation.
    return torch::from_blob((void*)raw_ptr, shape, strides, 0L, options);
}

void tensor_shape(at::Tensor t, bool explicit_nhwc, int& N, int& C, int& H, int& W)
{
    if (t.dim() == 3) {
	N = 1;
        if (explicit_nhwc) {
            C = t.size(2);
            H = t.size(0);
            W = t.size(1);
        } else {
	    C = t.size(0);
    	    H = t.size(1);
    	    W = t.size(2);
        }
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            N = t.size(0);
            C = t.size(3);
            H = t.size(1);
            W = t.size(2);
        } else {
            N = t.size(0);
            C = t.size(1);
            H = t.size(2);
            W = t.size(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

void tensor_strides(at::Tensor t, bool explicit_nhwc, int& stride_N, int& stride_C, int& stride_H, int& stride_W)
{
    if (t.dim() == 3) {
        if (explicit_nhwc) {
            stride_C = t.stride(2);
            stride_H = t.stride(0);
            stride_W = t.stride(1);
        } else {
	    stride_C = t.stride(0);
    	    stride_H = t.stride(1);
    	    stride_W = t.stride(2);
        }
        stride_N = t.size(0)*t.size(1)*t.size(2);
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            stride_N = t.stride(0);
            stride_C = t.stride(3);
            stride_H = t.stride(1);
            stride_W = t.stride(2);
        } else {
            stride_N = t.stride(0);
            stride_C = t.stride(1);
            stride_H = t.stride(2);
            stride_W = t.stride(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

template<class T>
inline __device__ void __zero(T* dst)
{
    *dst = T(0);
}

inline __device__ void __zero(int2* dst)
{
    *dst = {0, 0};
}

template<class T, bool channels_last, bool zero>
inline __device__ void push_pull_tensor(
        const T* __restrict__ data_in,  // local memory
        const int data_in_stride_C,
        const int data_in_stride_H,
        const int data_in_stride_W,
        int4* transfer_out,             // remote peer memory
        int4* transfer_in,              // local peer memory
	T* __restrict__ data_out,       // local memory
        const int data_out_stride_C,
        const int data_out_stride_H,
        const int data_out_stride_W,
	const int NC,
        const int NH,
        const int NW,
        const int thread_id,
        const int num_threads
	)
{
    const int count = NC*NH*NW;

    // communicate in 128b chunks
    // Note: NVLink flit size is 128b=16B. Use last 4B as a semaphore.
    static_assert(sizeof(T) <= 12);
    union Flit {
        T payload;
        uint uints[4];
    };

    // transfer buffers are contiguous
    int transfer_stride_C, transfer_stride_H, transfer_stride_W;
    if (channels_last) {
      transfer_stride_C = 1;
      transfer_stride_H = NC;
      transfer_stride_W = NW*NC;
    } else {
      transfer_stride_C = NH*HW;
      transfer_stride_H = NW;
      transfer_stride_W = 1;
    }

    // send data to peer GPU
    if (!zero) {
        for (int i = thread_id;  i < count;  i += num_threads) {
            // calculate position in buffers
            int c, h, w;
            if (channels_last) {
                const int j = i / NC;
                c = i % NC;
                h = j / NW;
                w = j % NW;
            } else {
                const int j = i / NW;
                w = i % NW;
                c = j / NH;
                h = j % NH;
            }
            const T& in = data_in[c*data_in_stride_C + h*data_in_stride_H + w*data_in_stride_W];
            int4& out = transfer_out[c*transfer_stride_C + h*transfer_stride_H + w*transfer_stride_W];

            // pack value into flit
            Flit flit;
            flit.payload = in;
            flit.uints[3] = 0xffffffff;

            // make sure peer buffer is ready
            // TODO Use double-buffering to allow push-only communication
            volatile int* ptr = reinterpret_cast<volatile int*>(&out);
            uint not_ready = 0xffffffff;
            do {
                uint r1, r2, r3;;
                asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" :
                             "=r"(r1),
                             "=r"(r2),
                             "=r"(r3),
                             "=r"(not_ready)
                             : "l"(ptr) : "memory");
            } while (not_ready != 0);

            // send flit to peer
            asm volatile("st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};" ::
                         "l"(ptr),
                         "r"(flit.uints[0]),
                         "r"(flit.uints[1]),
                         "r"(flit.uints[2]),
                         "r"(flit.uints[3])
                         : "memory");
        }
    }

    // recieve data from peer GPU
    for (int i = thread_id;  i < count;  i += num_threads) {
        // calculate position in buffers
        int c, h, w;
        if (channels_last) {
            const int j = i / NC;
            c = i % NC;
            h = j / NW;
            w = j % NW;
        } else {
            const int j = i / NW;
            w = i % NW;
            c = j / NH;
            h = j % NH;
        }
        int4& in = transfer_in[c*transfer_stride_C + h*transfer_stride_H + w*transfer_stride_W];
        T& out = data_out[c*data_out_stride_C + h*data_out_stride_H + w*data_out_stride_W];

        if (zero) {
	    __zero(&out);
        } else {
            // wait to recieve flit from peer
            Flit flit;
            volatile int* ptr = reinterpret_cast<volatile int*>(&in);
            do {
                asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" :
                             "=r"(flit.uints[0]),
                             "=r"(flit.uints[1]),
                             "=r"(flit.uints[2]),
                             "=r"(flit.uints[3])
                             : "l"(ptr) : "memory");
            } while (flit.uints[3] == 0);
            in = {0, 0, 0, 0};

            // unpack value from flit
            out = flit.payload;
        }
    }
}

template<class T, bool channels_last, bool top_zero, bool btm_zero>
#if __CUDA_ARCH__ >= 700
__launch_bounds__(THREADS_PER_CTA)
#endif
__global__ void push_pull_halos_1d_kernel(
        // top halo,
        const T* tih, int tih_stride_C, int tih_stride_H, int tih_stride_W,     // top input halo (local)
        T* tox,                                                                 // top output transfer buffer (remote peer)
        T* tix,                                                                 // top input transfer buffer (local peer)
        T* toh, int toh_stride_C, int toh_stride_H, int toh_stride_W,           // top output halo (local)
        // btm halo
        const T* bih, int bih_stride_C, int bih_stride_H, int bih_stride_W,     // btm input halo (local)
        T* box,                                                                 // btm output transfer buffer (remote peer)
        T* bix,                                                                 // btm input transfer buffer (local peer)
        T* boh, int boh_stride_C, int boh_stride_H, int boh_stride_W,           // btm output halo (local)
        // dimensions
        int NC, int NH, int NW
        )
{
    const int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int num_threads_per_side = (gridDim.x / 2) * blockDim.x;
    const bool in_top_block = thread_id < num_threads_per_side;
    const int side_thread_id = in_top_block ? thread_id : thread_id - num_threads_per_side;
    if (in_top_block) {
        push_pull_tensor<T,channels_last,top_zero>(
            tih, tih_stride_C, tih_stride_H, tih_stride_W,
            tox,
            tix,
            toh, toh_stride_C, toh_stride_H, toh_stride_W,
            NC, NH, NW,
            side_thread_id, num_threads_per_side);
    } else {
        push_pull_tensor<T,channels_last,btm_zero>(
            bih, bih_stride_C, bih_stride_H, bih_stride_W,
            box,
            bix,
            boh, boh_stride_C, boh_stride_H, boh_stride_W,
            NC, NH, NW,
            side_thread_id, num_threads_per_side);
    }
}

__global__ void delay_kernel(int delay_nanoseconds, int* counter)
{
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // waste time while doing something compiler can't predict, thus preventing it from optimizing away this code.
        int new_counter = 0;
        double elapsed = 0;
        clock_t start = clock();
        do {
            clock_t now = clock();
            elapsed = (double)(now - start)*1e9 / CLOCKS_PER_SEC;
            ++new_counter;
        } while (elapsed < (double)delay_nanoseconds);
        *counter = new_counter;
    }
}

}

namespace apex { namespace contrib { namespace peer_memory {

int64_t allocate_raw(int64_t size)
{
    float* ptr = 0L;
    hipMalloc(&ptr, size);
    hipMemset(ptr, 0, size);
    return (int64_t)ptr;
}

void free_raw(int64_t raw)
{
    hipFree((void*)raw);
}

void zero(int64_t raw, int64_t size)
{
    hipMemset((void*)raw, 0, size);
}

at::Tensor get_raw_ipc_address(int64_t raw)
{
    hipIpcMemHandle_t mem_handle;
    CUDACHECK( hipIpcGetMemHandle(&mem_handle, (void*)raw) );
    const int n = sizeof(hipIpcMemHandle_t);
    auto address_tensor = torch::empty({n}, torch::dtype(torch::kUInt8));
    auto address_tensor_p = address_tensor.data_ptr<uint8_t>();
    memcpy(address_tensor_p, (uint8_t*)&mem_handle, n);
    return address_tensor;
}

std::vector<int64_t> get_raw_peers(at::Tensor ipc_addresses, int peer_rank, int64_t raw)
{
    int peer_group_size = ipc_addresses.size(0);
    std::vector<int64_t> results(peer_group_size);
    for (int i = 0;  i < peer_group_size;  ++i) {
        if (i != peer_rank) {
            hipIpcMemHandle_t mem_handle;
            memcpy(&mem_handle, ipc_addresses.index({i}).data_ptr<uint8_t>(), sizeof(hipIpcMemHandle_t));
            void* p = 0L;
            CUDACHECK( hipIpcOpenMemHandle((void**)&p, mem_handle, hipIpcMemLazyEnablePeerAccess) );
            results[i] = (int64_t)p;
        } else {
            results[i] = (int64_t)raw;
        }
    }
    return results;
}

at::Tensor blob_view_half(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<at::Half>((at::Half*)raw, shape, torch::dtype(torch::kFloat16).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_float(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<float>((float*)raw, shape, torch::dtype(torch::kFloat32).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_int(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<int>((int*)raw, shape, torch::dtype(torch::kInt32).device(torch::kCUDA), channels_last);
}

void push_pull_halos_1d(
	bool diagnostics,
        bool explicit_nhwc,
        int numSM,                      // number of SMs to use (zero corresponds to all SMs)
	bool top_zero,			// if top halo should be zeroed
        at::Tensor top_in_halo,         // top input halo buffer (in local device memory, sent to top neighbor)
        at::Tensor top_out_transfer,    // top output transfer buffer (in top neighbor peer memory)
	at::Tensor top_in_transfer,	// top input transfer buffer (in local peer memory)
        at::Tensor top_out_halo,        // top output halo buffer (in local device memory, received from top neighbor)
	bool btm_zero,			// if btm halo should be zeroed
        at::Tensor btm_in_halo,         // btm input halo buffer (in local device memory, sent to btm neighbor)
        at::Tensor btm_out_transfer,    // btm output transfer buffer (in btm neighbor peer memory)
	at::Tensor btm_in_transfer,	// btm input transfer buffer (in local peer memory)
        at::Tensor btm_out_halo         // btm output halo buffer (in local device memory, received from btm neighbor)
        )
{
    // basic checks of inputs
    TORCH_CHECK(!(top_zero && btm_zero));
    TORCH_CHECK(top_in_halo.is_cuda());
    TORCH_CHECK(top_out_transfer.is_cuda());
    TORCH_CHECK(top_in_transfer.is_cuda());
    TORCH_CHECK(top_out_halo.is_cuda());
    TORCH_CHECK(btm_in_halo.is_cuda());
    TORCH_CHECK(btm_out_transfer.is_cuda());
    TORCH_CHECK(btm_in_transfer.is_cuda());
    TORCH_CHECK(btm_out_halo.is_cuda());

    // tensor shapes
    int tih_N, tih_C, tih_H, tih_W;
    tensor_shape(top_in_halo, explicit_nhwc, tih_N, tih_C, tih_H, tih_W);
    int toh_N, toh_C, toh_H, toh_W;
    tensor_shape(top_out_halo, explicit_nhwc, toh_N, toh_C, toh_H, toh_W);
    int bih_N, bih_C, bih_H, bih_W;
    tensor_shape(btm_in_halo, explicit_nhwc, bih_N, bih_C, bih_H, bih_W);
    int boh_N, boh_C, boh_H, boh_W;
    tensor_shape(btm_out_halo, explicit_nhwc, boh_N, boh_C, boh_H, boh_W);
    TORCH_CHECK(toh_N == tih_N && tih_N == boh_N && boh_N == bih_N &&
                toh_C == tih_C && tih_C == boh_C && boh_C == bih_C &&
                toh_H == tih_H && tih_H == boh_H && boh_H == bih_H &&
                toh_W == tih_W && tih_W == boh_W && boh_W == bih_W);
    int NN=toh_N, NC=toh_C, NH=toh_H, NW=toh_W;
    if (diagnostics) printf("NN=%d, NC=%d, NH=%d, NW=%d\n",NN,NC,NH,NW);
    TORCH_CHECK(NN == 1);

    // tensor strides
    int tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W;
    tensor_strides(top_in_halo, explicit_nhwc, tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W);
    int toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W;
    tensor_strides(top_out_halo, explicit_nhwc, toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W);
    int bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W;
    tensor_strides(btm_in_halo, explicit_nhwc, bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W);
    int boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W;
    tensor_strides(btm_out_halo, explicit_nhwc, boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W);

    // determine if nhwc
    bool is_nhwc = (toh_stride_C == 1);
    if (diagnostics) printf("is_nhwc = %s\n",is_nhwc?"true":"false");

    // peer memory buffers
    int tox_size = top_out_transfer.numel() * top_out_transfer.element_size();
    int tix_size = top_in_transfer.numel() * top_in_transfer.element_size();
    int box_size = btm_out_transfer.numel() * btm_out_transfer.element_size();
    int bix_size = btm_in_transfer.numel() * btm_in_transfer.element_size();
    if (!top_zero) {
        TORCH_CHECK(top_out_transfer.is_contiguous());
        TORCH_CHECK(top_in_transfer.is_contiguous());
    }
    if (!btm_zero) {
        TORCH_CHECK(btm_out_transfer.is_contiguous());
        TORCH_CHECK(btm_in_transfer.is_contiguous());
    }

    // figure out launch parameters
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    if (numSM <= 0 || numSM > prop.multiProcessorCount) {
      numSM = prop.multiProcessorCount;
    }
    auto current_stream = at::cuda::getCurrentCUDAStream();
    dim3 block(THREADS_PER_CTA,1,1);

    // helper macros to launch templated kernel
#define LAUNCH_PUSH_PULL_HALO_KERNEL_BASE(T, IS_HWC, TOP_ZERO, BTM_ZERO, KERNEL_ARGS, NUM_ELEMENTS) \
    do {                                                                \
        /* require 128b peer memory per element */                      \
        int peer_memory_size = NUM_ELEMENTS * 16;                       \
        if (!TOP_ZERO) {                                                \
            TORCH_CHECK(tox_size >= peer_memory_size && tix_size >= peer_memory_size); \
        }                                                               \
        if (!BTM_ZERO) {                                                \
            TORCH_CHECK(box_size >= peer_memory_size && bix_size >= peer_memory_size); \
        }                                                               \
                                                                        \
        /* launch kernel */                                             \
        int numBlocksPerSm;                                             \
        hipOccupancyMaxActiveBlocksPerMultiprocessor(                  \
            &numBlocksPerSm,                                            \
            push_pull_halos_1d_kernel<T,IS_HWC,TOP_ZERO,BTM_ZERO>,      \
            THREADS_PER_CTA,                                            \
            0);                                                         \
        dim3 grid(numSM*numBlocksPerSm,1,1);                            \
        if (grid.x % 2 != 0) {                                          \
            /* require even number of blocks (half for top, half for bottom) */ \
            grid.x -= 1;                                                \
        }                                                               \
        if ((grid.x / 2) * block.x > NUM_ELEMENTS) {                    \
            /* only need enough blocks to cover top and bottom halo elements */ \
            grid.x = 2 * ((NUM_ELEMENTS + block.x - 1) / block.x);      \
        }                                                               \
        hipLaunchCooperativeKernel(                                    \
            (void*)push_pull_halos_1d_kernel<T,IS_HWC,TOP_ZERO,BTM_ZERO>, \
            grid,                                                       \
            block,                                                      \
            KERNEL_ARGS,                                                \
            0,                                                          \
            current_stream);                                            \
    } while (false)
#define LAUNCH_PUSH_PULL_HALO_KERNEL(T, IS_HWC, KERNEL_ARGS, NUM_ELEMENTS) \
    do {                                                                \
        if (top_zero) {                                                 \
            LAUNCH_PUSH_PULL_HALO_KERNEL_BASE(T, IS_HWC, true, false, KERNEL_ARGS, NUM_ELEMENTS); \
        } else if (btm_zero) {                                          \
            LAUNCH_PUSH_PULL_HALO_KERNEL_BASE(T, IS_HWC, false, true, KERNEL_ARGS, NUM_ELEMENTS); \
        } else {                                                        \
            LAUNCH_PUSH_PULL_HALO_KERNEL_BASE(T, IS_HWC, false, false, KERNEL_ARGS, NUM_ELEMENTS); \
        }                                                               \
    } while (false)

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, top_out_halo.scalar_type(), "push_pull_halos_1d_kernel", [&]{
	if (diagnostics) printf("size(scalar_t) = %ld\n",sizeof(scalar_t));
        scalar_t* tih_p = top_inp_halo.data_ptr<scalar_t>();
        int4* tox_p = top_out_tx.data_ptr<int4>();
        int4* tix_p = top_inp_tx.data_ptr<int4>();
        scalar_t* toh_p = top_out_halo.data_ptr<scalar_t>();
        scalar_t* bih_p = btm_inp_halo.data_ptr<scalar_t>();
        int4* box_p = btm_out_tx.data_ptr<int4>();
        int4* bix_p = btm_inp_tx.data_ptr<int4>();
        scalar_t* boh_p = btm_out_halo.data_ptr<scalar_t>();
        if (diagnostics) printf("waypoint1\n");

        // do int2 vector loads if channel count permits
        int elem_size_in_bytes = toh_C * sizeof(scalar_t);
        int elem_size_in_int2 = (elem_size_in_bytes / 8);
        if (diagnostics) printf("elem_size_in_bytes = %d, elem_size_in_int2 = %d\n",elem_size_in_bytes,elem_size_in_int2);
        if (is_nhwc && elem_size_in_int2*8 == elem_size_in_bytes) {
            // can do int2 transfers
            int divisor = 8 / sizeof(scalar_t);
            if (diagnostics) printf("CAN DO INT2 :: divisor = %d\n",divisor);
            toh_stride_N /= divisor;   toh_stride_H /= divisor;    toh_stride_W /= divisor;
            tox_stride_N /= divisor;   tox_stride_H /= divisor;    tox_stride_W /= divisor;
            tix_stride_N /= divisor;   tix_stride_H /= divisor;    tix_stride_W /= divisor;
            tih_stride_N /= divisor;   tih_stride_H /= divisor;    tih_stride_W /= divisor;
            boh_stride_N /= divisor;   boh_stride_H /= divisor;    boh_stride_W /= divisor;
            box_stride_N /= divisor;   box_stride_H /= divisor;    box_stride_W /= divisor;
            bix_stride_N /= divisor;   bix_stride_H /= divisor;    bix_stride_W /= divisor;
            bih_stride_N /= divisor;   bih_stride_H /= divisor;    bih_stride_W /= divisor;
            NC /= divisor;
            if (diagnostics) {
                printf("divisor=%d\n",divisor);
                printf("tih_stride :: N=%d, C=%d, H=%d, W=%d\n",tih_stride_N,tih_stride_C,tih_stride_H,tih_stride_W);
                printf("toh_stride :: N=%d, C=%d, H=%d, W=%d\n",toh_stride_N,toh_stride_C,toh_stride_H,toh_stride_W);
                printf("bih_stride :: N=%d, C=%d, H=%d, W=%d\n",bih_stride_N,bih_stride_C,bih_stride_H,bih_stride_W);
                printf("boh_stride :: N=%d, C=%d, H=%d, W=%d\n",boh_stride_N,boh_stride_C,boh_stride_H,boh_stride_W);
                printf("NC=%d, NH=%d, NW=%d\n",NC,NH,NW);
            }
            void *kernel_args[] = {
                (int2**)&tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
                &tox_p,
                &tix_p,
                (int2**)&toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
                (int2**)&bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
                &box_p,
                &bix_p,
                (int2**)&boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
                &NC, &NH, &NW
            };
            int num_elem = NC*NH*NW;
            LAUNCH_PUSH_PULL_HALO_KERNEL(int2, true, kernel_args, num_elem);
        } else {
            // cannot do int2 transfers
            if (diagnostics) printf("CAN NOT DO INT2\n");
            void *kernel_args[] = {
		&tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
                &tox_p,
                &tix_p,
                &toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
                &bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
                &box_p,
                &bix_p,
                &boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
                &NC, &NH, &NW
            };
            int num_elem = NC*NH*NW;
            if (is_nhwc) {
                LAUNCH_PUSH_PULL_HALO_KERNEL(scalar_t, true, kernel_args, num_elem);
            } else {
                LAUNCH_PUSH_PULL_HALO_KERNEL(scalar_t, false, kernel_args, num_elem);
            }
        }
    } );

#undef LAUNCH_PUSH_PULL_HALO_KERNEL_BASE
#undef LAUNCH_PUSH_PULL_HALO_KERNEL
}

} } }
