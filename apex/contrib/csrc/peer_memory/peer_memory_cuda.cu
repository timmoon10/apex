#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <list>
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include "nccl.h"
namespace cg = cooperative_groups;

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if( err != hipSuccess ) {                        \
    char hostname[1024];                            \
    gethostname(hostname, 1024);                    \
    printf("%s: CUDA failure %s:%d '%s'\n",         \
         hostname,                                  \
        __FILE__,__LINE__,hipGetErrorString(err)); \
  }                                                 \
} while(0)

namespace {

/* Basic deleter function for from_blob function.
void deleter(void* ptr)
{
    printf("deleter(ptr=%p)\n",ptr);
    hipFree(ptr);
}
*/

template<class T>
at::Tensor blob_view(T* raw_ptr, std::vector<int64_t> shape, const at::TensorOptions& options, bool channels_last)
{
    size_t size = 1;
    std::vector<int64_t> strides(shape.size());
    if (channels_last) {
        assert(shape.size() == 4);
        strides[0] = shape[1]*shape[2]*shape[3];
        strides[1] = 1;
        strides[2] = shape[1]*shape[3];
        strides[3] = shape[1];
    } else {
        int idx = strides.size();
        for (auto it = shape.rbegin();  it != shape.rend();  ++it)
        {
	    strides[--idx] = size;
	    size *= *it;
        }
    }
    size *= sizeof(T);
    // TODO: Implement dynamic reuse of pooled peer memory.
    // We provide no deleter function because all peer memory allocations are static in this implementation.
    return torch::from_blob((void*)raw_ptr, shape, strides, 0L, options);
}

void tensor_shape(at::Tensor t, bool explicit_nhwc, int& N, int& C, int& H, int& W)
{
    if (t.dim() == 3) {
	N = 1;
        if (explicit_nhwc) {
            C = t.size(2);
            H = t.size(0);
            W = t.size(1);
        } else {
	    C = t.size(0);
    	    H = t.size(1);
    	    W = t.size(2);
        }
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            N = t.size(0);
            C = t.size(3);
            H = t.size(1);
            W = t.size(2);
        } else {
            N = t.size(0);
            C = t.size(1);
            H = t.size(2);
            W = t.size(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

void tensor_strides(at::Tensor t, bool explicit_nhwc, int& stride_N, int& stride_C, int& stride_H, int& stride_W)
{
    if (t.dim() == 3) {
        if (explicit_nhwc) {
            stride_C = t.stride(2);
            stride_H = t.stride(0);
            stride_W = t.stride(1);
        } else {
	    stride_C = t.stride(0);
    	    stride_H = t.stride(1);
    	    stride_W = t.stride(2);
        }
        stride_N = t.size(0)*t.size(1)*t.size(2);
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            stride_N = t.stride(0);
            stride_C = t.stride(3);
            stride_H = t.stride(1);
            stride_W = t.stride(2);
        } else {
            stride_N = t.stride(0);
            stride_C = t.stride(1);
            stride_H = t.stride(2);
            stride_W = t.stride(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

template<class T>
inline __device__ void __zero(T* dst)
{
    *dst = T(0);
}

inline __device__ void __zero(int4* dst)
{
    *dst = {0, 0, 0, 0};
}

template<class T, bool is_HWC, bool zero>
inline __device__ void strided_copy_kernel(
	T* __restrict__ dst,
        const int dst_stride_C,
        const int dst_stride_H,
        const int dst_stride_W,
	const T* __restrict__ src,
        const int src_stride_C,
        const int src_stride_H,
        const int src_stride_W,
	const int NC,
        const int NH,
        const int NW
	)
{
    const int tot_num_threads = gridDim.x * blockDim.x;
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int count = NC*NH*NW;
    for (int i = thread_id;  i < count;  i += tot_num_threads)
    {
	int c, h, w;
	if (is_HWC) {
	    w = i / NC;
	    c = i - w * NC;
	    h = w / NW;
	    w = w - h * NW;
	}
	else {
	    h = i / NW;
	    w = i - h * NW;
	    c = h / NH;
            h = h - c * NH;
	}
	int dst_off = c*dst_stride_C + h*dst_stride_H + w*dst_stride_W;
	if (zero) {
	    __zero(dst+dst_off);
	} else {
	    int src_off = c*src_stride_C + h*src_stride_H + w*src_stride_W;
	    dst[dst_off] = src[src_off];
	}
    }
}

// Waits until the first entry in an int4 flag is set or unset. Should
// only be called on main thread.
inline __device__ void wait_for_flag(volatile int* flag, bool wait_until_set)
{
    register int r1, r2, r3, r4;
    do {
        asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" : "=r"(r1), "=r"(r2), "=r"(r3), "=r"(r4) : "l"(flag) : "memory");
    } while (wait_until_set ^ (r1 != 0));
}

// Sets an int4 flag to {val, 0, 0, 0}. Should only be called on
// main thread.
inline __device__ void set_flag(volatile int* flag, const int val)
{
    register int r1{val}, r2{0}, r3{0}, r4{0};
    asm volatile("st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};" :: "l"(flag), "r"(r1), "r"(r2), "r"(r3), "r"(r4) : "memory");
}

template<class T, bool is_HWC, bool top_zero, bool btm_zero>
#if __CUDA_ARCH__ == 700 || __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 900
__launch_bounds__(128, 16)
#endif
__global__ void push_pull_halos_1d_kernel(
        // top halo,
        const T* toh, int toh_stride_C, int toh_stride_H, int toh_stride_W,     // top output halo
        T* tox, int tox_stride_C, int tox_stride_H, int tox_stride_W,           // top output tx buffer
        T* tix, int tix_stride_C, int tix_stride_H, int tix_stride_W,           // top input tx buffer
        T* tih, int tih_stride_C, int tih_stride_H, int tih_stride_W,           // top input halo
        // btm halo
        const T* boh, int boh_stride_C, int boh_stride_H, int boh_stride_W,     // btm output halo
        T* box, int box_stride_C, int box_stride_H, int box_stride_W,           // btm output tx buffer
        T* bix, int bix_stride_C, int bix_stride_H, int bix_stride_W,           // btm input tx buffer
        T* bih, int bih_stride_C, int bih_stride_H, int bih_stride_W,           // btm input halo
        // dimensions
        int NC, int NH, int NW,
        // signals
        int* tox_write_ready, int* tox_read_ready,
        int* tix_write_ready, int* tix_read_ready,
        int* box_write_ready, int* box_read_ready,
        int* bix_write_ready, int* bix_read_ready
        )
{
    const bool is_main_thread = blockIdx.x == 0 && threadIdx.x == 0;

    // wait until transfer buffers are ready
    if (is_main_thread) {
        if (!top_zero) {
            wait_for_flag(tox_write_ready, false);
            set_flag(tox_write_ready, -1);
        }
        if (!btm_zero) {
            wait_for_flag(box_write_ready, false);
            set_flag(box_write_ready, -1);
        }
    }
    cg::this_grid().sync();

    // push halos to transfer buffers
    if (!top_zero) {
        strided_copy_kernel<T,is_HWC,false>(tox, tox_stride_C, tox_stride_H, tox_stride_W,
                                            toh, toh_stride_C, toh_stride_H, toh_stride_W,
                                            NC, NH, NW);
    }
    if (!btm_zero) {
        strided_copy_kernel<T,is_HWC,false>(box, box_stride_C, box_stride_H, box_stride_W,
                                            boh, boh_stride_C, boh_stride_H, boh_stride_W,
                                            NC, NH, NW);
    }

    // synchronize with neighbors
    cg::this_grid().sync();
    if (is_main_thread) {
	__threadfence_system();
        if (!top_zero) set_flag(tox_read_ready, -1);
        if (!btm_zero) set_flag(box_read_ready, -1);
        if (!top_zero) {
            wait_for_flag(tix_read_ready, true);
            set_flag(tix_read_ready, 0);
        }
        if (!btm_zero) {
            wait_for_flag(bix_read_ready, true);
            set_flag(bix_read_ready, 0);
        }
    }
    cg::this_grid().sync();

    // pull halos from transfer buffers
    strided_copy_kernel<T,is_HWC,top_zero>(tih, tih_stride_C, tih_stride_H, tih_stride_W,
                                           tix, tix_stride_C, tix_stride_H, tix_stride_W,
                                           NC, NH, NW);
    strided_copy_kernel<T,is_HWC,btm_zero>(bih, bih_stride_C, bih_stride_H, bih_stride_W, bix,
                                           bix_stride_C, bix_stride_H, bix_stride_W,
                                           NC, NH, NW);

    // reset flags
    cg::this_grid().sync();
    if (is_main_thread) {
	__threadfence_system();
        if (!top_zero) set_flag(tix_write_ready, 0);
        if (!btm_zero) set_flag(bix_write_ready, 0);
    }
}

__global__ void delay_kernel(int delay_nanoseconds, int* counter)
{
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // waste time while doing something compiler can't predict, thus preventing it from optimizing away this code.
        int new_counter = 0;
        double elapsed = 0;
        clock_t start = clock();
        do {
            clock_t now = clock();
            elapsed = (double)(now - start)*1e9 / CLOCKS_PER_SEC;
            ++new_counter;
        } while (elapsed < (double)delay_nanoseconds);
        *counter = new_counter;
    }
}

}

namespace apex { namespace contrib { namespace peer_memory {

int64_t allocate_raw(int64_t size)
{
    float* ptr = 0L;
    hipMalloc(&ptr, size);
    hipMemset(ptr, 0, size);
    return (int64_t)ptr;
}

void free_raw(int64_t raw)
{
    hipFree((void*)raw);
}

void zero(int64_t raw, int64_t size)
{
    hipMemset((void*)raw, 0, size);
}

at::Tensor get_raw_ipc_address(int64_t raw)
{
    hipIpcMemHandle_t mem_handle;
    CUDACHECK( hipIpcGetMemHandle(&mem_handle, (void*)raw) );
    const int n = sizeof(hipIpcMemHandle_t);
    auto address_tensor = torch::empty({n}, torch::dtype(torch::kUInt8));
    auto address_tensor_p = address_tensor.data_ptr<uint8_t>();
    memcpy(address_tensor_p, (uint8_t*)&mem_handle, n);
    return address_tensor;
}

std::vector<int64_t> get_raw_peers(at::Tensor ipc_addresses, int peer_rank, int64_t raw)
{
    int peer_group_size = ipc_addresses.size(0);
    std::vector<int64_t> results(peer_group_size);
    for (int i = 0;  i < peer_group_size;  ++i) {
        if (i != peer_rank) {
            hipIpcMemHandle_t mem_handle;
            memcpy(&mem_handle, ipc_addresses.index({i}).data_ptr<uint8_t>(), sizeof(hipIpcMemHandle_t));
            void* p = 0L;
            CUDACHECK( hipIpcOpenMemHandle((void**)&p, mem_handle, hipIpcMemLazyEnablePeerAccess) );
            results[i] = (int64_t)p;
        } else {
            results[i] = (int64_t)raw;
        }
    }
    return results;
}

at::Tensor blob_view_half(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<at::Half>((at::Half*)raw, shape, torch::dtype(torch::kFloat16).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_float(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<float>((float*)raw, shape, torch::dtype(torch::kFloat32).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_int(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<int>((int*)raw, shape, torch::dtype(torch::kInt32).device(torch::kCUDA), channels_last);
}

void push_pull_halos_1d(
	bool diagnostics,
        bool explicit_nhwc,
        int numSM,                      // number of SMs to use
	bool top_zero,			// true if top halo should be zeroed
        at::Tensor top_out_halo,        // top output halo in sender device memory
        at::Tensor top_out_tx,          // top output transfer buffer in sender peer pool memory
	at::Tensor top_inp_tx,		// top input transfer buffer in top neighbor peer pool memory
        at::Tensor top_inp_halo,        // top input halo in receiver device memory
	bool btm_zero,			// true if btm halo should be zeroed
        at::Tensor btm_out_halo,        // btm output halo in sender device memory
        at::Tensor btm_out_tx,          // btm output transfer buffer in sender peer pool memory
	at::Tensor btm_inp_tx,		// btm input transfer buffer in btm neighbor peer pool memory
        at::Tensor btm_inp_halo,        // btm input halo in receiver device memory
        at::Tensor top_signal,          // top input signal in receiver device memory
        at::Tensor btm_signal,          // btm input signal in receiver device memory
        at::Tensor waits                // top and btm signals for this rank
        )
{
    // basic checks of inputs
    TORCH_CHECK(top_out_halo.is_cuda());
    TORCH_CHECK(top_out_tx.is_cuda());
    TORCH_CHECK(top_inp_tx.is_cuda());
    TORCH_CHECK(top_inp_halo.is_cuda());
    TORCH_CHECK(btm_out_halo.is_cuda());
    TORCH_CHECK(btm_out_tx.is_cuda());
    TORCH_CHECK(btm_inp_tx.is_cuda());
    TORCH_CHECK(btm_inp_halo.is_cuda());
    TORCH_CHECK(top_signal.is_cuda());
    TORCH_CHECK(btm_signal.is_cuda());
    TORCH_CHECK(waits.is_cuda());
    TORCH_CHECK(!(top_zero && btm_zero));

    // shapes and strides
    int toh_N, toh_C, toh_H, toh_W;
    tensor_shape(top_out_halo, explicit_nhwc, toh_N, toh_C, toh_H, toh_W);
    int tox_N, tox_C, tox_H, tox_W;
    tensor_shape(top_out_tx, explicit_nhwc, tox_N, tox_C, tox_H, tox_W);
    int tix_N, tix_C, tix_H, tix_W;
    tensor_shape(top_inp_tx, explicit_nhwc, tix_N, tix_C, tix_H, tix_W);
    int tih_N, tih_C, tih_H, tih_W;
    tensor_shape(top_inp_halo, explicit_nhwc, tih_N, tih_C, tih_H, tih_W);
    TORCH_CHECK(
            (toh_N == tox_N && tox_N == tix_N && tix_N == tih_N) &&
            (toh_C == tox_C && tox_C == tix_C && tix_C == tih_C) &&
            (toh_H == tox_H && tox_H == tix_H && tix_H == tih_H) &&
            (toh_W == tox_W && tox_W == tix_W && tix_W == tih_W));
    int boh_N, boh_C, boh_H, boh_W;
    tensor_shape(btm_out_halo, explicit_nhwc, boh_N, boh_C, boh_H, boh_W);
    int box_N, box_C, box_H, box_W;
    tensor_shape(btm_out_tx, explicit_nhwc, box_N, box_C, box_H, box_W);
    int bix_N, bix_C, bix_H, bix_W;
    tensor_shape(btm_inp_tx, explicit_nhwc, bix_N, bix_C, bix_H, bix_W);
    int bih_N, bih_C, bih_H, bih_W;
    tensor_shape(btm_inp_halo, explicit_nhwc, bih_N, bih_C, bih_H, bih_W);
    TORCH_CHECK(
            (boh_N == box_N && box_N == bix_N && bix_N == bih_N) &&
            (boh_C == box_C && box_C == bix_C && bix_C == bih_C) &&
            (boh_H == box_H && box_H == bix_H && bix_H == bih_H) &&
            (boh_W == box_W && box_W == bix_W && bix_W == bih_W));
    TORCH_CHECK(
	    (toh_N == boh_N) &&
	    (toh_C == boh_C) &&
	    (toh_H == boh_H) &&
	    (toh_W == boh_W));
    int NC=toh_C, NH=toh_H, NW=toh_W;
    if (diagnostics) printf("NC=%d, NH=%d, NW=%d\n",NC,NH,NW);

    int toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W;
    tensor_strides(top_out_halo, explicit_nhwc, toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W);
    int tox_stride_N, tox_stride_C, tox_stride_H, tox_stride_W;
    tensor_strides(top_out_tx, explicit_nhwc, tox_stride_N, tox_stride_C, tox_stride_H, tox_stride_W);
    int tix_stride_N, tix_stride_C, tix_stride_H, tix_stride_W;
    tensor_strides(top_inp_tx, explicit_nhwc, tix_stride_N, tix_stride_C, tix_stride_H, tix_stride_W);
    int tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W;
    tensor_strides(top_inp_halo, explicit_nhwc, tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W);
    int boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W;
    tensor_strides(btm_out_halo, explicit_nhwc, boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W);
    int box_stride_N, box_stride_C, box_stride_H, box_stride_W;
    tensor_strides(btm_out_tx, explicit_nhwc, box_stride_N, box_stride_C, box_stride_H, box_stride_W);
    int bix_stride_N, bix_stride_C, bix_stride_H, bix_stride_W;
    tensor_strides(btm_inp_tx, explicit_nhwc, bix_stride_N, bix_stride_C, bix_stride_H, bix_stride_W);
    int bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W;
    tensor_strides(btm_inp_halo, explicit_nhwc, bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W);

    // determine if nhwc
    auto is_nhwc = (toh_stride_C == 1) ? true : false;
    if (diagnostics) printf("is_nhwc = %s\n",is_nhwc?"true":"false");

    // figure out launch parameters
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    assert(numSM > 0 && numSM <= prop.multiProcessorCount);
    auto current_stream = at::cuda::getCurrentCUDAStream();
    const int numThreads = 128;
    dim3 block(numThreads,1,1);
    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, top_out_halo.scalar_type(), "push_pull_halos_1d_kernel", [&]{
	    if (diagnostics) printf("size(scalar_t) = %ld\n",sizeof(scalar_t));
            scalar_t* toh_p = top_out_halo.data_ptr<scalar_t>();
            scalar_t* tox_p = top_out_tx.data_ptr<scalar_t>();
            scalar_t* tix_p = top_inp_tx.data_ptr<scalar_t>();
            scalar_t* tih_p = top_inp_halo.data_ptr<scalar_t>();
            scalar_t* boh_p = btm_out_halo.data_ptr<scalar_t>();
            scalar_t* box_p = btm_out_tx.data_ptr<scalar_t>();
            scalar_t* bix_p = btm_inp_tx.data_ptr<scalar_t>();
            scalar_t* bih_p = btm_inp_halo.data_ptr<scalar_t>();
	    if (diagnostics) printf("waypoint1\n");
            int* tox_write_ready = waits.data_ptr<int>();
            int* tox_read_ready = top_signal.data_ptr<int>() + 12; // bix_read_ready in neighbor
            int* tix_write_ready = top_signal.data_ptr<int>() + 8; // box_write_ready in neighbor
            int* tix_read_ready = waits.data_ptr<int>() + 4;
            int* box_write_ready = waits.data_ptr<int>() + 8;
            int* box_read_ready = btm_signal.data_ptr<int>() + 4; // tix_read_ready in neighbor
            int* bix_write_ready = btm_signal.data_ptr<int>(); // tox_write_ready in neighbor
            int* bix_read_ready = waits.data_ptr<int>() + 12;
	    if (diagnostics) printf("waypoint2\n");

            // do int4 vector loads if channel count permits
            int elem_size_in_bytes = toh_C * sizeof(scalar_t);
            int elem_size_in_int4 = (elem_size_in_bytes / 16);
	    if (diagnostics) printf("elem_size_in_bytes = %d, elem_size_in_int4 = %d\n",elem_size_in_bytes,elem_size_in_int4);
            if (is_nhwc && elem_size_in_int4*16 == elem_size_in_bytes) {
                // can do int4 transfers
	        int divisor = toh_C / elem_size_in_int4;
		if (diagnostics) printf("CAN DO INT4 :: divisor = %d\n",divisor);
		toh_stride_N /= divisor;   toh_stride_H /= divisor;    toh_stride_W /= divisor;
		tox_stride_N /= divisor;   tox_stride_H /= divisor;    tox_stride_W /= divisor;
		tix_stride_N /= divisor;   tix_stride_H /= divisor;    tix_stride_W /= divisor;
		tih_stride_N /= divisor;   tih_stride_H /= divisor;    tih_stride_W /= divisor;
		boh_stride_N /= divisor;   boh_stride_H /= divisor;    boh_stride_W /= divisor;
		box_stride_N /= divisor;   box_stride_H /= divisor;    box_stride_W /= divisor;
		bix_stride_N /= divisor;   bix_stride_H /= divisor;    bix_stride_W /= divisor;
		bih_stride_N /= divisor;   bih_stride_H /= divisor;    bih_stride_W /= divisor;
		NC /= divisor;
		if (diagnostics) {
                    printf("divisor=%d\n",divisor);
                    printf("toh_stride :: N=%d, C=%d, H=%d, W=%d\n",toh_stride_N,toh_stride_C,toh_stride_H,toh_stride_W);
                    printf("tox_stride :: N=%d, C=%d, H=%d, W=%d\n",tox_stride_N,tox_stride_C,tox_stride_H,tox_stride_W);
                    printf("tix_stride :: N=%d, C=%d, H=%d, W=%d\n",tix_stride_N,tix_stride_C,tix_stride_H,tix_stride_W);
                    printf("tih_stride :: N=%d, C=%d, H=%d, W=%d\n",tih_stride_N,tih_stride_C,tih_stride_H,tih_stride_W);
                    printf("boh_stride :: N=%d, C=%d, H=%d, W=%d\n",boh_stride_N,boh_stride_C,boh_stride_H,boh_stride_W);
                    printf("box_stride :: N=%d, C=%d, H=%d, W=%d\n",box_stride_N,box_stride_C,box_stride_H,box_stride_W);
                    printf("bix_stride :: N=%d, C=%d, H=%d, W=%d\n",bix_stride_N,bix_stride_C,bix_stride_H,bix_stride_W);
                    printf("bih_stride :: N=%d, C=%d, H=%d, W=%d\n",bih_stride_N,bih_stride_C,bih_stride_H,bih_stride_W);
                    printf("NC=%d, NH=%d, NW=%d\n",NC,NH,NW);
                }
		void *kernelArgs[] = {
		    (int4**)&toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
		    (int4**)&tox_p, &tox_stride_C, &tox_stride_H, &tox_stride_W,
		    (int4**)&tix_p, &tix_stride_C, &tix_stride_H, &tix_stride_W,
		    (int4**)&tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
		    (int4**)&boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
		    (int4**)&box_p, &box_stride_C, &box_stride_H, &box_stride_W,
		    (int4**)&bix_p, &bix_stride_C, &bix_stride_H, &bix_stride_W,
		    (int4**)&bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
		    &NC, &NH, &NW,
                    &tox_write_ready, &tox_read_ready,
                    &tix_write_ready, &tix_read_ready,
                    &box_write_ready, &box_read_ready,
                    &bix_write_ready, &bix_read_ready
		};
		if (top_zero) {
		    int numBlocksPerSm;
		    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<int4,true,true,false>, numThreads, 0);
		    dim3 grid(numSM*numBlocksPerSm,1,1);
		    hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<int4,true,true,false>, grid, block, kernelArgs, 0, current_stream);
		} else if (btm_zero) {
		    int numBlocksPerSm;
		    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<int4,true,false,true>, numThreads, 0);
		    dim3 grid(numSM*numBlocksPerSm,1,1);
		    hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<int4,true,false,true>, grid, block, kernelArgs, 0, current_stream);
		} else {
		    int numBlocksPerSm;
		    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<int4,true,false,false>, numThreads, 0);
		    dim3 grid(numSM*numBlocksPerSm,1,1);
		    hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<int4,true,false,false>, grid, block, kernelArgs, 0, current_stream);
		}
            } else {
                // cannot do int4 transfers
		if (diagnostics) printf("CAN NOT DO INT4\n");
		void *kernelArgs[] = {
		    &toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
		    &tox_p, &tox_stride_C, &tox_stride_H, &tox_stride_W,
		    &tix_p, &tix_stride_C, &tix_stride_H, &tix_stride_W,
		    &tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
		    &boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
		    &box_p, &box_stride_C, &box_stride_H, &box_stride_W,
		    &bix_p, &bix_stride_C, &bix_stride_H, &bix_stride_W,
		    &bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
		    &NC, &NH, &NW,
                    &tox_write_ready, &tox_read_ready,
                    &tix_write_ready, &tix_read_ready,
                    &box_write_ready, &box_read_ready,
                    &bix_write_ready, &bix_read_ready
		};
                int numBlocksPerSm;
                if (is_nhwc) {
		    if (top_zero) {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,true,true,false>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,true,true,false>, grid, block, kernelArgs, 0, current_stream);
		    } else if (btm_zero) {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,true,false,true>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,true,false,true>, grid, block, kernelArgs, 0, current_stream);
		    } else {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,true,false,false>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,true,false,false>, grid, block, kernelArgs, 0, current_stream);
		    }
                } else {
		    if (top_zero) {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,false,true,false>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,false,true,false>, grid, block, kernelArgs, 0, current_stream);
		    } else if (btm_zero) {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,false,false,true>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,false,false,true>, grid, block, kernelArgs, 0, current_stream);
		    } else {
			hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,false,false,false>, numThreads, 0);
			dim3 grid(numSM*numBlocksPerSm,1,1);
			hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,false,false,false>, grid, block, kernelArgs, 0, current_stream);
		    }
                }
	    }
        } );
}

} } }
